//1. Write a CUDA program to print the message “Hello World” and demonstrate threads by varying BLOCK_WIDTH to different sizes.


#include <hip/hip_runtime.h>
#include <stdio.h>
#define NUM_BLOCKS 32
#define BLOCK_WIDTH 3
__global__ void hello()
{
printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
//printf("Hello world! I'm thread %d\n", threadIdx.x);
}int main(int argc,char **argv)
{
// launch the kernel
hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();
// force the printf()s to flush
hipDeviceSynchronize();
printf("That's all!\n");
return 0;
}